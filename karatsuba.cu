
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void karatsuba(uint64_t *X, uint64_t *Y, uint64_t *result) {
    uint64_t x = *X;
    uint64_t y = *Y;

    printf("x = %llu; y = %llu\n", x, y);

    if (x == 0 || y == 0){
      *result = 0;
      return;
    }

    if (x < 2 && y < 2) {
        *result = x & y;
        return;
    }

    int bitsX = __log2f((float)x) + 1;
    int bitsY = __log2f((float)y) + 1;

    int maxBits = max(bitsX, bitsY);
    int mid = maxBits >> 1;

    // Splitting the numbers
    uint64_t XH = x >> mid;
    uint64_t XL = x & ((1 << mid) - 1);
    uint64_t YH = y >> mid;
    uint64_t YL = y & ((1 << mid) - 1);

    uint64_t *A, *B, *C;
    hipMalloc(&A, sizeof(uint64_t));
    hipMalloc(&B, sizeof(uint64_t));
    hipMalloc(&C, sizeof(uint64_t));

    uint64_t *XH_d, *YH_d, *XL_d, *YL_d;
    hipMalloc(&XH_d, sizeof(uint64_t));
    hipMalloc(&YH_d, sizeof(uint64_t));
    hipMalloc(&XL_d, sizeof(uint64_t));
    hipMalloc(&YL_d, sizeof(uint64_t));

    *XH_d = XH;
    *YH_d = YH;
    *XL_d = XL;
    *YL_d = YL;

    karatsuba<<<1, 1>>>(XH_d, YH_d, A);  // A = XH * YH
    karatsuba<<<1, 1>>>(XL_d, YL_d, B);  // B = XL * YL

    uint64_t XH_plus_XL = XH + XL;
    uint64_t YH_plus_YL = YH + YL;

    uint64_t *XH_plus_XL_d, *YH_plus_YL_d;
    hipMalloc(&XH_plus_XL_d, sizeof(uint64_t));
    hipMalloc(&YH_plus_YL_d, sizeof(uint64_t));

    *XH_plus_XL_d = XH_plus_XL;
    *YH_plus_YL_d = YH_plus_YL;

    karatsuba<<<1, 1>>>(XH_plus_XL_d, YH_plus_YL_d, C);  // C = (XH + XL) * (YH + YL)

    uint64_t resA = *A; 
    uint64_t resB = *B;
    uint64_t resC = *C;

    uint64_t D = resC - resA - resB;
    *result = (resA << (mid << 1)) + (D << mid) + resB;

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(XH_d);
    hipFree(YH_d);
    hipFree(XL_d);
    hipFree(YL_d);
    hipFree(XH_plus_XL_d);
    hipFree(YH_plus_YL_d);
}

int main() {
    printf("Parallel Karatsuba from Kumar: \n");
    uint64_t X = 3;
    uint64_t Y = 2;
    uint64_t result = 0;

    uint64_t *X_d, *Y_d, *result_d;

    hipMalloc((void**)&X_d, sizeof(uint64_t));
    hipMalloc((void**)&Y_d, sizeof(uint64_t));
    hipMalloc((void**)&result_d, sizeof(uint64_t));

    hipMemcpy(X_d, &X, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(Y_d, &Y, sizeof(uint64_t), hipMemcpyHostToDevice);

    karatsuba<<<1, 1>>>(X_d, Y_d, result_d);

    hipDeviceSynchronize(); 

    hipMemcpy(&result, result_d, sizeof(uint64_t), hipMemcpyDeviceToHost);

    printf("Karatsuba Result: %llu\n", result);

    hipFree(X_d);
    hipFree(Y_d);
    hipFree(result_d);

    return 0;
}
