#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define DIGITS 10 
#define BLOCK_WIDTH 16 

__global__ void bmulTiled(size_t *Aglb, size_t *Bglb, uint64_t *Cglb) {
    __shared__ size_t Ash[BLOCK_WIDTH], Bsh[BLOCK_WIDTH]; 
    __shared__ uint64_t Csh[2 * BLOCK_WIDTH];

    int ii = blockIdx.y * BLOCK_WIDTH, i = threadIdx.y; // 0 <= i < T 
    int jj = blockIdx.x * BLOCK_WIDTH, j = threadIdx.x; // 0 <= j < T
    
    // copy A and B from global to shared memory & initialize Csh
    if (threadIdx.y == 0) {
        Ash[j] = Aglb[ii + j]; 
        Bsh[j] = Bglb[jj + j]; 
        Csh[j] = 0;
        Csh[j + BLOCK_WIDTH] = 0;
    }
    __syncthreads();

    if (ii + jj + i + j < DIGITS) {
        uint64_t prod = ((uint64_t)Ash[i]) * ((uint64_t)Bsh[j]);
        atomicAdd(&Csh[i + j], prod); // atomic in shared memory
    }
    __syncthreads();

    int tid = i * BLOCK_WIDTH + j;
    if (tid < 2 * BLOCK_WIDTH && ii + jj + tid < 2 * DIGITS) {  // atomic in global memory 
      atomicAdd(&Cglb[ii + jj + tid], Csh[tid]); 
    }
}

__global__ void carryPropagation(uint64_t *Cglb, size_t *Result, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= size) return;

    uint64_t carry = 0;
    for (int i = 0; i < size; i++) {
        uint64_t temp = Cglb[i] + carry;
        Result[i] = temp % 10; 
        carry = temp / 10;      
    }

    if (carry > 0 && size < 2 * DIGITS) { 
        Result[size] = carry;
    }
}

void printArray(size_t *arr, int size) {
    bool leadingZero = true;
    for (int i = size - 1; i >= 0; i--) {
        if (arr[i] != 0) leadingZero = false;
        if (!leadingZero) printf("%llu", arr[i]);
    }
    if (leadingZero) printf("0");
    printf("\n");
}

int main() {
    printf("Tiled Quadratic Multiplication: ");
    size_t h_A[DIGITS] = {5, 4, 3, 2, 1}; // 12345 in reverse
    size_t h_B[DIGITS] = {6, 5, 4, 3, 2}; // 23456 in reverse
    size_t h_Result[2 * DIGITS] = {0};  

    size_t *d_A, *d_B;
    uint64_t *d_C;
    size_t *d_Result;

    hipMalloc((void **)&d_A, DIGITS * sizeof(size_t));
    hipMalloc((void **)&d_B, DIGITS * sizeof(size_t));
    hipMalloc((void **)&d_C, 2 * DIGITS * sizeof(uint64_t));
    hipMalloc((void **)&d_Result, 2 * DIGITS * sizeof(size_t));

    hipMemcpy(d_A, h_A, DIGITS * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DIGITS * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, 2 * DIGITS * sizeof(uint64_t));
    hipMemset(d_Result, 0, 2 * DIGITS * sizeof(size_t));

    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 dimGrid((DIGITS + BLOCK_WIDTH) / BLOCK_WIDTH, (DIGITS + BLOCK_WIDTH - 1) / BLOCK_WIDTH);

    printArray(h_A, DIGITS);
    printArray(h_B, DIGITS);

    bmulTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();

    carryPropagation<<<(DIGITS + 255) / 256, 256>>>(d_C, d_Result, 2 * DIGITS);
    hipMemcpy(h_Result, d_Result, 2 * DIGITS * sizeof(size_t), hipMemcpyDeviceToHost);

    printArray(h_Result, 2 * DIGITS);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_Result);

    return 0;
  }
